
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 512
#define NUM_BLOCKS 16

__global__ void histo_kernel(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins)
{
     __shared__ unsigned int shared_bins[4096];
    int tid = threadIdx.x;
    for (int i = tid; i < num_bins; i += blockDim.x) {
        shared_bins[i] = 0;
    }

    __syncthreads();
    
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = gid; i < num_elements; i += stride) {
        unsigned int bin = input[i];
        if (bin < num_bins) {
            atomicAdd(&shared_bins[bin], 1);
        }
    }

    __syncthreads();

    for (int i = tid; i < num_bins; i += blockDim.x) {
        atomicAdd(&bins[i], shared_bins[i]);
    }
    
}

void histogram(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins) {

	  histo_kernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(input, bins, num_elements, num_bins);

}


